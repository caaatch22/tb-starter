
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <algorithm>
#include <chrono>
#include <immintrin.h>
#include <hipblas.h>

class Timer {
public:
 Timer() noexcept : start(std::chrono::high_resolution_clock::now()) {}

 [[nodiscard]] auto elapsed() const noexcept -> std::chrono::duration<double> {
   const auto now = std::chrono::high_resolution_clock::now();
   return std::chrono::duration<double>(now - start);
 }

 void reset() noexcept { start = std::chrono::high_resolution_clock::now(); }

private:
 std::chrono::time_point<std::chrono::high_resolution_clock> start;
};

static std::random_device seed_random_device;
static std::mt19937 engine(seed_random_device());
static std::mt19937_64 engine64(seed_random_device());

inline static double rng_double() {
  return static_cast<double>(engine()) / engine.max();
}

class Matrix {
public:
 Matrix(size_t m, size_t n)
     : m_(m), n_(n), data_(new(std::align_val_t(64)) float[m * n]) {}
 ~Matrix() { delete[] data_; }

 Matrix(Matrix const& rhs)
     : m_(rhs.m_), n_(rhs.n_), data_(new float[m_ * n_]) {
   std::copy(rhs.data_, rhs.data_ + m_ * n_, data_);
 }
 Matrix(Matrix&& rhs) noexcept : m_(rhs.m_), n_(rhs.n_), data_(rhs.data_) {}
 [[nodiscard]] bool operator==(const Matrix& rhs) const noexcept {
   return m_ == rhs.m_ && n_ == rhs.n_ &&
          std::equal(data_, data_ + m_ * n_, rhs.data_,
                     [](float a, float b) { return std::abs(a - b) < 1e-6; });
 }

 [[nodiscard]] bool operator!=(const Matrix& rhs) const noexcept { return !(*this == rhs); }

 Matrix& operator=(Matrix const& rhs) {
   if (this != &rhs) {
     Matrix tmp(rhs);
     std::swap(m_, tmp.m_);
     std::swap(n_, tmp.n_);
     std::swap(data_, tmp.data_);
   }
   return *this;
 }

 Matrix& operator=(Matrix&& rhs) {
   std::swap(m_, rhs.m_);
   std::swap(n_, rhs.n_);
   std::swap(data_, rhs.data_);
   return *this;
 }


 [[nodiscard]] float const* data() const noexcept { return data_; }
 [[nodiscard]] float* data() noexcept { return data_; }
 [[nodiscard]] size_t row() const noexcept { return m_; }
 [[nodiscard]] size_t col() const noexcept { return n_; }

 void zeros() {
   // #pragma omp simd
   for (size_t i = 0; i < m_ * n_; i++) {
     data_[i] = 0.0f;
   }
 }
 void ones() { std::fill(data_, data_ + m_ * n_, 1.0f); }
 void random() { std::generate(data_, data_ + m_ * n_, rng_double); }

 [[nodiscard]] float& operator()(size_t i, size_t j) {
   return data_[i * n_ + j];
 }

 [[nodiscard]] float const& operator()(size_t i, size_t j) const {
   return data_[i * n_ + j];
 }

 [[nodiscard]] float& operator()(size_t i) { return data_[i]; }
 [[nodiscard]] float const& operator()(size_t i) const { return data_[i]; }

 Matrix make_aligned(size_t const block_size) const {
   auto const rows = ((row() + block_size - 1) / block_size) * block_size;
   auto const cols = ((col() + block_size - 1) / block_size) * block_size;
   Matrix result(rows, cols);
   for (size_t i = 0; i < row(); i++) {
     for (size_t j = 0; j < col(); j++) {
       result(i, j) = (*this)(i, j);
     }
   }
   return result;
 }

 void from_aligned(Matrix const& src) {
   for (size_t i = 0; i < row(); i++) {
     for (size_t j = 0; j < col(); j++) {
       (*this)(i, j) = src(i, j);
     }
   }
 }

private:
 size_t m_;
 size_t n_;
 alignas(64) float* data_;
};

void gemm(Matrix& C, Matrix const& A, Matrix const& B) {
  constexpr static size_t BLOCK_SIZE = 64;
  alignas(64) static float localA[BLOCK_SIZE][BLOCK_SIZE];
  alignas(64) static float localB[BLOCK_SIZE][BLOCK_SIZE];
  alignas(64) static float localC[BLOCK_SIZE][BLOCK_SIZE];
  
#pragma omp threadprivate(localA, localB, localC)
  size_t const matSize = A.row();
  size_t const blockNum = matSize / BLOCK_SIZE;
    // Traverse blocks.
#pragma omp parallel for
    for (size_t bi = 0; bi < blockNum; bi++) {
      for (size_t bj = 0; bj < blockNum; bj++) {
        // Clear localC.
        for (size_t i = 0; i < BLOCK_SIZE; i++) {
#pragma omp simd
          for (size_t j = 0; j < BLOCK_SIZE; j ++) {
              localC[i][j] = 0.0f;
          }
        }
  
        for (size_t bk = 0; bk < blockNum; bk++) {
          // Copy local block.
          for (size_t i = 0; i < BLOCK_SIZE; i++) {
            for (size_t j = 0; j < BLOCK_SIZE; j++) {
              size_t const aIdx = bi * BLOCK_SIZE * blockNum * BLOCK_SIZE +
                            i * blockNum * BLOCK_SIZE + bk * BLOCK_SIZE + j;
              size_t const bIdx = bk * BLOCK_SIZE * blockNum * BLOCK_SIZE +
                            i * blockNum * BLOCK_SIZE + bj * BLOCK_SIZE + j;
              localA[i][j] = A(aIdx);
              localB[i][j] = B(bIdx);
            }
          }
  
        //   Block GEMM.
                  for (size_t i = 0; i < BLOCK_SIZE; i++) {
                    for (size_t k = 0; k < BLOCK_SIZE; k++) {
          #pragma omp simd
                      for (size_t j = 0; j < BLOCK_SIZE; j++) {
                        localC[i][j] += localA[i][k] * localB[k][j];
                      }
                    }
                  }
        }
        // Copy localC back.
        for (size_t i = 0; i < BLOCK_SIZE; i++) {
  #pragma omp simd
          for (size_t j = 0; j < BLOCK_SIZE; j++) {
            size_t cIdx = bi * BLOCK_SIZE * blockNum * BLOCK_SIZE +
                          i * blockNum * BLOCK_SIZE + bj * BLOCK_SIZE + j;
            C(cIdx) = localC[i][j];
          }
        }
      }
    }
  }
  

double GFlops(size_t m, size_t n, size_t k, double seconds) {
  return 2.0 * m * n * k / (1e9 * seconds);
}

#define TILE_WIDTH 64
__global__ void matrixMulKernel(float* M, float* N, float* P, int Width) {
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float value = 0.0f;
    for (size_t i = 0; i < Width / TILE_WIDTH; i ++) {
        Mds[ty][tx] = M[row * Width + i * TILE_WIDTH + tx];
        Nds[ty][tx] = M[(i * TILE_WIDTH + ty) * Width + col];
        __syncthreads();

        for (size_t k = 0; k < TILE_WIDTH; k ++) {
            value += Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads();
    }
    P[row * Width + col] = value;
}

void check_equal(Matrix const& A, Matrix const& B) {
    if (A != B) {
      for (size_t i = 0; i < 3; i++) {
        for (size_t j = 0; j < 3; j++) {
            std::cout << A(i, j) << " ";
            std::cout << B(i, j) << std::endl;
        }
      }
      throw std::runtime_error("A != B");
    }
  }

int main() {
  constexpr int N = 1024 * 8;
  Matrix A(N, N);
  Matrix B(N, N);
  Matrix C(N, N);
  A.random();
  B.random();
  Timer timer;
  gemm(C, A, B);
  auto time = timer.elapsed();

  std::cout << "Time: " << time.count() << " s" << std::endl;
  std::cout << "GFlops: " << GFlops(N, N, N, time.count()) << std::endl;
  for (size_t i = 0; i < 2; i++) {
    for (size_t j = 0; j < 2; j++) {
      std::cout << C(i, j) << " ";
    }
    std::cout << std::endl;
  }

  Matrix C2(N, N);

  float* d_A, *d_B, *d_C;
  hipMalloc(&d_A, N * N * sizeof(float));
  hipMalloc(&d_B, N * N * sizeof(float));
  hipMalloc(&d_C, N * N * sizeof(float));

  timer.reset();
  hipMemcpy(d_A, A.data(), N * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B.data(), N * N * sizeof(float), hipMemcpyHostToDevice);
  dim3 dimGrid(N / TILE_WIDTH, N / TILE_WIDTH, 1);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
  matrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);
  hipMemcpy(C2.data(), d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
  time = timer.elapsed();
  std::cout << "Time: " << time.count() << " s" << std::endl;
  std::cout << "GFlops: " << GFlops(N, N, N, time.count()) << std::endl;
//   check_equal(C, C2);
  return 0;
}